#include <hip/hip_runtime.h>
#include <stdio.h>

void initialInt(int *ip, int size){
  for(int i = 0; i<size; i++){
    ip[i] = i;
  }
}

void printMatrix(int *C, const int nx, const int ny){
  int *ic = C;
  printf("\n Matrix: (%d, %d) \n", nx,  ny);
  for (int iy = 0; iy < ny; iy++){
    for(int ix = 0; ix < nx; ix++){
      printf("%3d", ic[ix]);
    }
    ic += nx;
    printf("\n");
  }
  printf("\n");
}

__global__ void printfThreadIndex(int *A, const int nx, const int ny){
  int ix = threadIdx.x + blockIdx.x * blockDim.x;
  int iy = threadIdx.y + blockIdx.x * blockDim.y;

  unsigned int idx = iy*nx + ix;

  printf("thread_id (%d,%d) block_id (%d, %d) coordinate (%d, %d) global index %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv){
  printf("%s Starting...\n", argv[0]);

  //get device information
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Using Device %d:%s\n", dev, deviceProp.name);
  hipSetDevice(dev);

  //set matrix dimention
  int nx = 8;
  int ny = 6;
  int nxy = nx*ny;
  int nBytes = nxy * sizeof(float);

  //malloc host memory
  int *h_A;
  h_A = (int *)malloc(nBytes);

  //initialize host matrix with integer
  initialInt(h_A, nxy);
  printMatrix(h_A, nx, ny);

  //malloc device memory
  int *d_MatA;
  hipMalloc((void **)&d_MatA, nBytes);

  //transfer data from host to device
  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

  //setup execution configuration
  dim3 block(4, 2);
  dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

  //invoke the kernel
  printfThreadIndex<<< grid, block >>>(d_MatA, nx, ny);
  hipDeviceSynchronize();

  // free host and device memory
  hipFree(d_MatA);
  free(h_A);

  //reset device
  hipDeviceReset();

  return 0;
}
