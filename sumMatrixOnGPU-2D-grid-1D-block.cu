#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "initialData.h"
#include "checkResult.h"
#include "cpuSecond.h"

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny){
  float *ia = A;
  float *ib = B;
  float *ic = C;

  for(int iy=0; iy<ny; iy++){
    for(int ix=0; ix<nx; ix++){
      ic[ix] = ia[ix]+ib[ix];
    }
    ia += nx; ib += nx; ic += nx;
  }
}

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny){
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = blockIdx.y;
  unsigned int idx = iy*nx + ix;
  if(ix < nx){
    MatC[idx] = MatA[idx] + MatB[idx];
  }
}
int main(int argc, char **argv){
  printf("%s Starting...\n", argv[0]);

  //get device information
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Using Device %d:%s\n", dev, deviceProp.name);
  hipSetDevice(dev);

  //set up data size of matrix
  int nx = 1<<14;
  int ny = 1<<14;

  int nxy = nx*ny;
  int nBytes = nxy*sizeof(float);
  printf("Matrix size: nx %d ny %d \n", nx, ny);

  //malloc host memory
  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  //initialize data at host side
  double iStart = cpuSecond();
  initialData(h_A, nxy);
  initialData(h_B, nxy);
  double iElaps = cpuSecond() - iStart;

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  //add matrix at host side for result checks
  iStart = cpuSecond();
  sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
  iElaps = cpuSecond() - iStart;

  //malloc device global memory
  float *d_MatA, *d_MatB, *d_MatC;
  hipMalloc((void **)&d_MatA, nBytes);
  hipMalloc((void **)&d_MatB, nBytes);
  hipMalloc((void **)&d_MatC, nBytes);

  //transfer data from host to device
  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

  //invoke kernel at host side
  int dimx = 256;
  dim3 block(dimx, 1);
  dim3 grid((nx+block.x-1)/block.x , ny);

  iStart = cpuSecond();
  sumMatrixOnGPUMix<<< grid, block >>>(d_MatA, d_MatB, d_MatC, nx, ny);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("sumMatrixOnGPUMix<<< (%d, %d), (%d, %d) >>> elapsed %f sec\n", grid.x, grid.y, block.x, block.y, iElaps);

  //copy kernel result back to host side
  hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

  //check device results
  checkResult(hostRef, gpuRef, nxy);

  //free device global memory
  hipFree(d_MatA);
  hipFree(d_MatB);
  hipFree(d_MatC);

  //free host memory
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  //reset device
  hipDeviceReset();
  return 0;
}
