#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

void initialData(float* ip, int size){
  time_t t;
  srand((unsigned int)time(&t));
  for(int i = 0; i<size; i++){
    ip[i] = (float)(rand() & 0xFF)/10.0f;
  }
}

void checkResult(float *hostRef, float *gpuRef, const int N){
  double epsilon = 1.0E-8;
  bool match = 1;
  for(int i = 0; i<N; i++){
    if(abs(hostRef[i] - gpuRef[i]) > epsilon){
      match = 0;
      printf("Arrays do not match!\n");
      printf("Host %5.2f GPU %5.2f at current %d", hostRef[i], gpuRef[i], i);
      break;
    }
  }
  if(match) printf("Arrays match. \n \n");
}

double cpuSecond(){
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void sumArraysOnHost(float* A, float* B, float* C, const int N){
  for(int idx = 0; idx<N; idx++){
    C[idx] = A[idx] + B[idx];
  }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N) C[i] = A[i] + B[i];
}


int main(int argc, char **argv){
  printf("%s Starting... \n", argv[0]);

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Using Device %d:%s\n", dev, deviceProp.name);
  hipSetDevice(dev);

  // set up date size of vectors
  int nElem = 1<<24;
  printf("Vector size %d\n", nElem);

  // malloc host memory
  size_t nBytes = nElem * sizeof(float);

  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A     = (float *)malloc(nBytes);
  h_B     = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef  = (float *)malloc(nBytes);

  double iStart, iElaps;

  // initialize data at host side
  iStart = cpuSecond();
  initialData(h_A, nElem);
  initialData(h_B, nElem);

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  // add vector at host side for result checks

  iStart = cpuSecond();
  sumArraysOnHost(h_A, h_B, hostRef, nElem);
  iElaps = cpuSecond() - iStart;

  // malloc device global memory
  float *d_A, *d_B, *d_C;
  hipMalloc((float **)&d_A, nBytes);
  hipMalloc((float **)&d_B, nBytes);
  hipMalloc((float **)&d_C, nBytes);

  // transfer data from host to device
  hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

  // invoke kernel at host side
  int iLen = 1024;
  dim3 block(iLen);
  dim3 grid((nElem+block.x-1)/block.x);

  iStart = cpuSecond();
  sumArraysOnGPU<<< grid, block >>>(d_A, d_B, d_C, nElem);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("sumArraysOnGPU <<< %d, %d >>> Time elapsed %f sec\n", grid.x, block.x, iElaps);

  // copy kernel result back to host side
  hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

  // check device results
  checkResult(hostRef, gpuRef, nElem);

  // free device global memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // free host memory
  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  return 0;
}
